#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "helper.h"

void checkErrorCuda(hipError_t cudaStatus)
{
    if (cudaStatus != hipSuccess)
        {
            printf("Cuda Error: %s\n", hipGetErrorString(cudaStatus));

            exit(1);
        }
}

void checkErrorMemory(void* val)
{
    if (val == NULL)
        {
            printf("Error allocating memory in CudaHelper\n");
            exit(1);
        }
}


__global__ void calculateMatching(int *d_pictureMat, int* pictureDim, int *d_objectMat, int* objectDim, double* d_matchingThreshold, int* d_indexFound, int* d_flag)
{
    int globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int limit = ((*pictureDim) - (*objectDim) + 1) * ((*pictureDim) - (*objectDim) + 1) ;

    if (globalThreadIndex < limit) {
        int rowIndex = globalThreadIndex / ((*pictureDim) - (*objectDim) + 1);    
        int colIndex = globalThreadIndex % ((*pictureDim) - (*objectDim) + 1);   
        int offset = rowIndex * (*pictureDim) + colIndex;   
        double match = 0;
        
        for (int i = 0; i < (*objectDim); i++){
            for(int  j = 0; j < (*objectDim); j++){
                int pictureVal = *(d_pictureMat + offset + i*(*pictureDim) + j);   
                int objectVal = *(d_objectMat + i*(*objectDim) + j);
                match += abs((pictureVal - objectVal) / pictureVal);
            }
        }

        if (match / ((*objectDim) * (*objectDim)) <  *d_matchingThreshold)
        {
            *d_flag = 1;
            *(d_indexFound + globalThreadIndex * 2 ) = rowIndex;
            *(d_indexFound + globalThreadIndex * 2 + 1) = colIndex;
        }
    }
}

__host__ void calculateMatchingOnGPU(Picture *picture, Object *object, double *matchingThreshold, int* foundFlag, int* indexsFound, int size)
{
    int* d_flag;
    checkErrorCuda(hipMalloc((void **)&d_flag, sizeof(int)));
    checkErrorCuda(hipMemcpy(d_flag, foundFlag, sizeof(int), hipMemcpyHostToDevice));

    // allocate memory for indexs match array 
    int* d_indexsFound;
    checkErrorCuda(hipMalloc((void**)&d_indexsFound, 2 * size * sizeof(int)));
    checkErrorCuda(hipMemcpy(d_indexsFound, indexsFound, 2 * size * sizeof(int), hipMemcpyHostToDevice));


    // allocate memory for the result on the GPU
    double* d_matchingThreshold;
    checkErrorCuda(hipMalloc((void **)&d_matchingThreshold, sizeof(double)));
    checkErrorCuda(hipMemcpy(d_matchingThreshold, matchingThreshold, sizeof(double), hipMemcpyHostToDevice));

    // allocate memory for the picture dimension on the GPU and copy the picture dimension to the GPU
    int* d_pictureDim;
    checkErrorCuda(hipMalloc((void **)&d_pictureDim, sizeof(int)));
    checkErrorCuda(hipMemcpy(d_pictureDim, &picture->dim, sizeof(int), hipMemcpyHostToDevice));

    // allocate memory for the object dimension on the GPU and copy the object dimension to the GPU
    int* d_objectDim;
    checkErrorCuda(hipMalloc((void **)&d_objectDim, sizeof(int)));
    checkErrorCuda(hipMemcpy(d_objectDim, &object->dim, sizeof(int), hipMemcpyHostToDevice));

    // allocate memory for the picture colors matrix on the GPU and copy the picture colors matrix to the GPU
    int* d_pictureMat;
    checkErrorCuda(hipMalloc((void **)&d_pictureMat, picture->dim * picture->dim * sizeof(int)));
    checkErrorCuda(hipMemcpy(d_pictureMat, picture->mat, picture->dim * picture->dim * sizeof(int), hipMemcpyHostToDevice));

    // allocate memory for the object sub colors matrix on the GPU and copy the object sub colors matrix to the GPU
    int* d_objectMat;
    checkErrorCuda(hipMalloc((void **)&d_objectMat, object->dim * object->dim * sizeof(int)));
    checkErrorCuda(hipMemcpy(d_objectMat, object->mat, object->dim * object->dim * sizeof(int), hipMemcpyHostToDevice));


    int blockSize = 512;                                    // threads per block
    int numBlocks = (size + blockSize - 1) / blockSize;     // block per grid
    
    // call the kernel function 
    calculateMatching <<<numBlocks, blockSize >>> (d_pictureMat, d_pictureDim, d_objectMat, d_objectDim, d_matchingThreshold, d_indexsFound, d_flag);

    checkErrorCuda(hipDeviceSynchronize());

    // check if the kernel function was called successfully
    checkErrorCuda(hipGetLastError());

    // copy the matched indexs to CPU
    checkErrorCuda(hipMemcpy(indexsFound, d_indexsFound, 2 * size * sizeof(int), hipMemcpyDeviceToHost));

    // copy found object flag
    checkErrorCuda(hipMemcpy(foundFlag, d_flag, sizeof(int), hipMemcpyDeviceToHost));    
    if(*foundFlag == 1)
    {
        (*picture).objFound++;
    }
    // free the memory on the GPU
    hipFree(d_matchingThreshold);
    hipFree(d_pictureMat);
    hipFree(d_objectMat);
    hipFree(d_pictureDim);
    hipFree(d_objectDim);
    hipFree(d_indexsFound);

}

